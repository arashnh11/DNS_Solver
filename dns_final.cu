#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <math.h>
#include <time.h>
#include <vector>
#include <iomanip>
#include <algorithm>
#include <string>
#include <map>
#include <stdint.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/reduce.h>

using namespace std;

#define BLOCK_SIZE 32 // Number of threads in x and y direction - Maximum Number of threads per block = 32 * 32 = 1024

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void Temperature_solver(int nx, int ny, int wu, int wv, int wT, float dx, float dy, float dt, float Re, float Pr, float *u, float *v, float *Told, float *T)
{

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if (i > 0 && i < nx && j > 0 && j < ny){

		Told[i * wT + j] = T[i * wT + j];
		T[i * wT + j] = T[i * wT + j] + dt*(-0.5*(u[i * wu + j] + u[(i - 1) * wu + j])*(1.0 / (2.0*dx)*(T[(i + 1) * wT + j] - T[(i - 1) * wT + j])) - 0.5*(v[i * wv + j] + v[i * wv + j - 1])*(1.0 / (2.0*dy)*(T[i * wT + j + 1] - T[i * wT + j - 1])) + 1 / (Re*Pr)*(1 / pow(dx, 2.0f)*(T[(i + 1) * wT + j] - 2.0*T[i * wT + j] + T[(i - 1) * wT + j]) + 1 / pow(dy, 2.0f)*(T[i * wT + j + 1] - 2 * T[i * wT + j] + T[i * wT + j - 1])));
	}
	__syncthreads();
}



__global__ void PressureSolve(float * p_d, const float * p_old, float * abs_d, const float * us_d, const float * vs_d, int p_xlength, int p_ylength, int wp, int wu, int wv, float dx, float dy, float dt)
{

	int i = threadIdx.x + blockDim.x*blockIdx.x;
	int j = threadIdx.y + blockDim.y*blockIdx.y;


	if (i > 0 && i < p_xlength && j > 0 && j < p_ylength)
	{
		//        __syncthreads();

		p_d[i * wp + j] = pow(dx, 2.0f)*pow(dy, 2.0f) / (-2.0*(pow(dx, 2.0f) + pow(dy, 2.0f)))*(-1.0 / pow(dx, 2.0f)*(p_old[(i + 1) * wp + j] + p_old[(i - 1) * wp + j] + p_old[i * wp + j + 1] + p_old[i * wp + j - 1]) + 1.0 / dt*(1.0 / dx*(us_d[i * wu + j] - us_d[(i - 1) * wu + j]) + 1.0 / dy*(vs_d[i * wv + j] - vs_d[i * wv + j - 1])));
		__syncthreads();

		abs_d[i * wp + j] = p_d[i * wp + j] - p_old[i * wp + j];

		__syncthreads();

		abs_d[i * wp + j] = abs_d[i * wp + j] * abs_d[i * wp + j];
		//__syncthreads();
	} // end if


} // end global


__global__ void PressureBC(float * p_d, float * p_ref, int nx, int ny, float dy, int wp)
{

	int i = threadIdx.x + blockDim.x*blockIdx.x;
	int j = threadIdx.y + blockDim.y*blockIdx.y;

	if (i >= 0 && i < nx + 1 && j == 0){
		p_d[i * wp + j] = p_ref[i * wp + j + 1]; // bottom wall - Final
	}
	__syncthreads();
	if (i >= 0 && i < nx + 1 && j == ny){
		p_d[i * wp + j] = p_ref[i * wp + j - 1]; // Upper - no flux
	}
	__syncthreads();
	if (j >= 0 && j < ny + 1 && i == 0){
		p_d[i * wp + j] = p_ref[(i + 1) * wp + j]; // left wall - not the inlet - Final
	}
	__syncthreads();
	if (j >= 0 && j < ny + 1 && i == nx && j*dy < 2.0){
		p_d[i * wp + j] = p_ref[(i - 1) * wp + j]; // right wall - not the outlet - Final

		// printf("POSITIVE ");
	}
	__syncthreads();
	if (j >= 0 && j < ny + 1 && i == nx && j*dy >= 2.0){
		p_d[i * wp + j] = -p_ref[(i - 1) * wp + j]; // pressure outlet - static pressure is zero - Final
		// printf("NEGATIVE ");    

	}
	//__syncthreads();

}


int main()
{
	try
	{
		// output format
		float start_clock = clock();
		ofstream f("result_gpu.txt"); // Solution Results
		f.setf(ios::fixed | ios::showpoint);
		f << setprecision(5);

		ofstream g("convergence_gpu.txt"); // Convergence history
		g.setf(ios::fixed | ios::showpoint);
		g << setprecision(5);
		cout.setf(ios::fixed | ios::showpoint);
		cout << setprecision(5);

		//ofstream file_p_before("p_before_gpu_BC.txt");
		//file_p_before.setf(ios::fixed | ios::showpoint);
		//file_p_before << setprecision(3); 
		//ofstream file_p_after("p_after_gpu_BC.txt");
		//file_p_after.setf(ios::fixed | ios::showpoint);
		//file_p_after << setprecision(3);

		// Input parameters 
		float Re, Pr, Fr, T_L, T_0, T_amb, dx, dy, t, eps, /* beta, */ iter, maxiter, tf, st, counter, column, u_wind, T_R, Lx, Ly;
		Lx = 4.0; Ly = 5.0; // Domain dimensions
		int ni = 10.0; // Number of nodes per unit length in x direction
		int nj = 10.0; // Number of nodes per unit length in y direction
		int nx = Lx * ni; int ny = Ly * nj; // Number of Nodes in each direction
		u_wind = 1; // Reference velocity
		st = 0.00005 * 2; // Total variance criteria
		eps = 0.001; // Pressure convergence criteria
		tf = 100; // Final time step
		Pr = 0.5*(0.709 + 0.711); // Prandtl number
		Re = 250.0; Fr = 0.3; // Non-dimensional numbers for inflow conditions
		dx = Lx / (nx - 1); dy = Ly / (ny - 1); // dx and dy
		//beta = 1; // Successive over relaxation factor (SOR)
		t = 0; // Initial time step
		T_L = 100.0; // Left wall temperature (C)
		T_R = 50.0; // Right wall temperature (C)
		T_amb = 25.0; // Ambient air temperature (C)
		T_0 = 50.0; // Initial air temperature
		T_L = T_L + 273.15; T_0 = T_0 + 273.15; T_amb = T_amb + 273.15; T_R = T_R + 273.15;// Unit conversion to (K)
		maxiter = 500; // Maximum iteration at each time step
		counter = 0; // initial row for output monitoring
		column = 1; // Column number for output display

		// Records number of clicks a step takes
		std::map<string, uint32_t> stepTimingAccumulator;

		// Host Vectors

		thrust::host_vector<float> u(nx * (ny + 1));
		thrust::host_vector<float> us(nx*(ny + 1));
		thrust::host_vector<float> uold(nx * (ny + 1));
		int wu = ny + 1;

		thrust::host_vector<float> v((nx + 1) * ny);
		thrust::host_vector<float> vs((nx + 1) * ny);
		thrust::host_vector<float> vold((nx + 1) * ny);
		int wv = ny;

		thrust::host_vector<float> p((nx + 1) * (ny + 1));
		//   thrust::host_vector<float> abs((nx + 1) * (ny + 1));
		int wp = ny + 1;


		thrust::host_vector<float> T((nx + 1) * (ny + 1));
		int wT = ny + 1;

		thrust::host_vector<float> Told((nx + 1) * (ny + 1));
		thrust::host_vector<float> om(nx * ny);
		thrust::host_vector<float> vc(nx * ny);
		thrust::host_vector<float> uc(nx * ny);
		thrust::host_vector<float> pc(nx * ny);
		thrust::host_vector<float> Tc(nx*ny);
		// thrust::host_vector<float> abs_h((nx+1) * (ny + 1));
		int wc = ny;

		hipFree(0);
		thrust::device_vector<float> us_d(nx*(ny + 1));
		thrust::device_vector<float> vs_d((nx + 1) * ny);
		thrust::device_vector<float> p_d((nx + 1) * (ny + 1), 0);
		thrust::device_vector<float> p_old((nx + 1) * (ny + 1), 0);
		thrust::device_vector<float> p_ref((nx + 1) * (ny + 1));
		thrust::device_vector<float> abs_d((nx + 1) * (ny + 1));
		gpuErrchk( hipPeekAtLastError() );
		// Time step size stability criterion

		float mt1 = 0.25*pow(dx, 2.0) / (1.0 / Re); float Rer = 1.0 / Re; float mt2 = 0.25*pow(dy, 2.0) / (1.0 / Re);
		float dt;

		if (mt1 > Rer)
		{
			dt = Rer;
		}
		else
		{
			dt = mt1;
		}

		if (dt > mt2)
		{
			dt = mt2;
		}


		//......................................................................................
		// Step 0 - It can be parallelized
		// Initializing the flow variable (Temperature)  
		// Boundary conditions for T (Initialization)
		int step0_start = clock();
		for (int i = 0; i < nx + 1; i++)
		{
			for (int j = 0; j < ny + 1; j++)
			{
				T[i * wT + j] = T_0 / T_amb;
			} // end for j
		} // end for i
		//......................................................................................
		int step0_end = clock();
		stepTimingAccumulator["Step 0, Initializing Temperature"] += step0_end - step0_start;
		//......................................................................................

		// Marching in Time - Outermost loop

		while (t <= tf)
		{

			iter = 0;

			int stepi1_start = clock();
			//........................................................................................
			// Step i1 - it can be parallelized 
			// boundary conditions for u velocity

			for (int i = 0; i < nx; i++)
			{
				for (int j = 0; j < ny + 1; j++)
				{
					if (i == 0 && j > 0 && j < ny)
					{
						if (j*dy < 2.0)
						{
							u[i * wu + j] = 0; // left wall - Final
						}
						else
						{
							u[i * wu + j] = u_wind; // left inlet - Final
						}
					}
					else if (i == nx - 1 && j>0 && j < ny)
					{
						if (j*dy < 2.0)
						{
							u[i * wu + j] = 0; // Right wall has 0 horizontal velocity - Final
						}
						else
						{
							u[i * wu + j] = u[(i - 1) * wu + j]; // right outlet - no velocity change
						}
					}
					else if (j == 0)
					{
						u[i * wu + j] = -u[i * wu + j + 1]; // bottom ghost - Final
					}
					else if (j == ny)
					{
						u[i * wu + j] = u[i * wu + j - 1]; // upper ghost - Final
					}
				} // end for j
			} // end for i
			int stepi1_end = clock();
			stepTimingAccumulator["Step i1 - Set Horizontal Velocity Boundary Conditions"] += stepi1_end - stepi1_start;
			//...............................................................................................


			//.........................................................................................
			// Step i2 - it can be parallelized
			// boundary conditions for v velocity
			int stepi2_start = clock();

			for (int i = 0; i < nx + 1; i++)
			{
				for (int j = 0; j < ny; j++)
				{
					if (j == 0 && i > 0 && i < nx)
					{
						v[i * wv + j] = 0; // bottom wall - Final
					}
					else if (j == ny - 1 && i > 0 && i < nx)
					{
						v[i * wv + j] = v[i * wv + j - 1]; // upper wall - Final
					}
					else if (i == 0)
					{
						v[i * wv + j] = -v[(i + 1) * wv + j]; // left ghost (Left Wall and inlet has 0 vertical velocity) - Final
					}
					else if (i == nx)
					{
						if (j*dy < 2.0)
						{
							v[i * wv + j] = -v[(i - 1) * wv + j]; // right ghost (Right wall has 0 vertical velocity) - Final
						}
						else
						{
							v[i * wv + j] = v[(i - 1) * wv + j]; // right outlet - no velocity gradient
						}
					}
				} // end for j
			} // end for I
			int stepi2_end = clock();
			stepTimingAccumulator["Step i2 - Set Vertical Velocity Boundary Conditions"] += stepi2_end - stepi2_start;
			//...............................................................................................

			//...............................................................................................
			int step1_start = clock();
			//.........................................................................................
			// Step 1 - it can be parallelized - Solve for intermediate velocity values

			// u - us - vh - a 

			for (int i = 1; i < nx - 1; i++)
			{
				for (int j = 1; j < ny; j++)
				{
					float vh = 1.0 / 4.0*(v[i * wv + j] + v[(i + 1) * wv + j] + v[i * wv + j - 1] + v[(i + 1) * wv + j - 1]); // v hat
					float a = u[i * wu + j] * 1.0 / (2.0*dx)*(u[(i + 1) * wu + j] - u[(i - 1) * wu + j]) + vh*1.0 / (2.0*dy)*(u[i * wu + j + 1] - u[i * wu + j - 1]); // a
					us[i * wu + j] = dt / Re*(1.0 / pow(dx, 2.0)*(u[(i + 1) * wu + j] - 2.0*u[i * wu + j] + u[(i - 1) * wu + j]) + 1.0 / pow(dy, 2.0)*(u[i * wu + j + 1] - 2.0*u[i * wu + j] + u[i * wu + j - 1])) - a*dt + u[i * wu + j]; // u star
				} // end for j
			} // end for i

			//..........................................................................................
			// Step 1 - it can be parallelized
			// v - vs - uh - b
			for (int i = 1; i < nx; i++)
			{
				for (int j = 1; j < ny - 1; j++)
				{
					float uh = 1.0 / 4.0*(u[i * wu + j] + u[i * wu + j + 1] + u[(i - 1) * wu + j] + u[(i - 1) * wu + j + 1]);
					float b = uh*1.0 / (2.0*dx)*(v[(i + 1) * wv + j] - v[(i - 1) * wv + j]) + v[i * wv + j] * 1.0 / (2.0*dy)*(v[i * wv + j + 1] - v[i * wv + j - 1]); // b
					vs[i * wv + j] = dt / Re*(1.0 / pow(dx, 2.0)*(v[(i + 1) * wv + j] - 2.0*v[i * wv + j] + v[(i - 1) * wv + j]) + 1.0 / pow(dy, 2.0)*(v[i * wv + j + 1] - 2.0*v[i * wv + j] + v[i * wv + j - 1])) + dt / pow(Fr, 2.0)*(0.5*(T[i * wT + j] + T[i * wT + j - 1]) - 1) / (0.5*(T[i * wT + j] + T[i * wT + j - 1])) - b*dt + v[i * wv + j]; // v 
				} // end for j
			} // end for i

			//...........................................................................................
			// vs and us on Boundary conditions

			for (int i = 0; i < nx; i++)
			{
				us[i * wu + 0] = -us[i * wu + 1]; // bottom ghost - Final
			} // end for j

			//...........................................................................................
			for (int j = 0; j < ny + 1; j++)
			{
				if (j*dy < 2.0)
				{
					us[0 * wu + j] = 0; // left wall - FInal
					us[(nx - 1) * wu + j] = 0; // right wall - Final
				}
				else
				{
					us[0 * wu + j] = u_wind; // left inlet - Final
				}
			}
			//...........................................................................................

			for (int j = 0; j < ny; j++)
			{
				vs[0 * wv + j] = -vs[1 * wv + j]; // left ghost (Both wall and inlet have 0 vs) - Final
				if (j*dy < 2.0)
				{
					vs[nx * wv + j] = -vs[(nx - 1) * wv + j]; // right ghost (Only the right wall - Final
				}
				else
				{
					vs[nx * wv + j] = vs[(nx - 1) * wv + j]; // right outlet - no flux
				}
			}
			//............................................................................................

			for (int i = 0; i < nx + 1; i++)
			{
				vs[i * wv + 0] = 0; // Bottom wall - Final
			} // end for i
			//............................................................................................

			int step1_end = clock();
			stepTimingAccumulator["Step 1 - Solve for intermediate velocities"] += step1_end - step1_start;


			//...............................................................................................
			// Step 2 - Parallel GPU version
			// Poisson equation for pressure

			int step2_start = clock();

			// Cuda set up
			int p_xlength = nx;
			int p_ylength = ny;

			float *ptr_us = thrust::raw_pointer_cast(&us_d[0]);
			float *ptr_vs = thrust::raw_pointer_cast(&vs_d[0]);
			float *ptr_p = thrust::raw_pointer_cast(&p_d[0]);
			float *ptr_p_old = thrust::raw_pointer_cast(&p_old[0]);
			float *ptr_abs = thrust::raw_pointer_cast(&abs_d[0]);
			float *ptr_p_ref = thrust::raw_pointer_cast(&p_ref[0]);

			float error = 1.0; iter = 0;
			// float diffp = 0;
			us_d = us;
			vs_d = vs;
			cout << t << endl;
			// Begin Jacobi loop
			while (error > eps){
				gpuErrchk( hipPeekAtLastError() );
				//error = 0.0;
				//  p_d = p;
				p_old = p_d;

				// SOR pressure solver
				PressureSolve<<< dim3( (ny+1)/BLOCK_SIZE + 1, (nx+1)/BLOCK_SIZE + 1, 1) , dim3(BLOCK_SIZE,BLOCK_SIZE,1)>>>(ptr_p, ptr_p_old, ptr_abs, ptr_us, ptr_vs, p_xlength, p_ylength, wp, wu, wv, dx, dy, dt);
				hipDeviceSynchronize();
				//	p = p_d;
				p_ref = p_d;

				error = thrust::reduce(abs_d.begin(), abs_d.end());

				/*  	    for (int i = 1; i < nx; i++)
				{
				for (int j = 1; j < ny; j++)
				{
				diffp = pow((p[i * wp + j] - p_old[i * wp + j]), 2.0);
				error = error + diffp;
				} // end for j
				} // end for i
				*/

				/* for(int i = 0; i < nx + 1; ++i)
				{
				for(int j = 0; j < ny + 1; ++j)
				{
				file_p_before << p[i * wp + j] << "\t";
				}
				file_p_before << endl;
				}
				*/
				// Apply boundary conditions

				PressureBC<<< dim3( (ny+1)/BLOCK_SIZE + 1, (nx+1)/BLOCK_SIZE + 1, 1) , dim3(BLOCK_SIZE,BLOCK_SIZE,1)>>>(ptr_p, ptr_p_ref, nx, ny, dy, wp);

				hipDeviceSynchronize();
				// p = p_d;
				//file_p_after << p.size() << endl;


				/* for(int i = 0; i < nx + 1; ++i)
				{
				for(int j = 0; j < ny + 1; ++j)
				{
				file_p_after << p[i * wp + j] << "\t";
				}
				file_p_after << endl;
				} */

				error = pow(error, 0.5);
				iter = iter + 1;
				if (iter > maxiter){
					break;
				}

			} // end while eps

			p = p_d;
			/*          
			break;
			error = pow(error, 0.5);
			iter = iter + 1;            
			if (iter == maxiter){
			break;
			}


			} // end while eps
			*/
			int step2_end = clock();
			stepTimingAccumulator["Step 2 - Solve for pressure until tolerance or max iterations"] += step2_end - step2_start;


			//.................................................................................................
			// Step 3 - It can be parallelized 
			// velocity update - projection method
			int step3_start = clock();

			// u

			for (int i = 1; i < nx - 1; i++)
			{
				for (int j = 1; j < ny; j++)
				{
					uold[i * wu + j] = u[i * wu + j];
					u[i * wu + j] = us[i * wu + j] - dt / dx*(p[(i + 1) * wp + j] - p[i * wp + j]);
				} // end for j
			} // end for i
			//................................................

			// v

			for (int i = 1; i < nx; i++)
			{
				for (int j = 1; j < ny - 1; j++)
				{
					vold[i * wv + j] = v[i * wv + j];
					v[i * wv + j] = vs[i * wv + j] - dt / dy*(p[i * wp + j + 1] - p[i * wp + j]);
				} // end for j
			} // end for i
			int step3_end = clock();
			stepTimingAccumulator["Step 3 - Velocity Update"] += step3_end - step3_start;
			//...............................................................................................

			//...............................................................................................
			// Step 4 - It can be parallelized
			// Solving for temperature
			int step4_start = clock();

			thrust::device_vector<float> d_T = T;
			thrust::device_vector<float> d_Told = Told;
			thrust::device_vector<float> d_u = u;
			thrust::device_vector<float> d_v = v;

			int gridsize_x = nx/BLOCK_SIZE + 1;
			int gridsize_y = ny/BLOCK_SIZE + 1;

			dim3 dimgrid(gridsize_x, gridsize_y, 1); // The grid has #gridsize blocks in x and 1 block in y and 1 block in z direction
			dim3 dimblock(BLOCK_SIZE, BLOCK_SIZE, 1);

			float *ptr_u = thrust::raw_pointer_cast(&d_u[0]);
			float *ptr_v = thrust::raw_pointer_cast(&d_v[0]);
			float *ptr_T = thrust::raw_pointer_cast(&d_T[0]);
			float *ptr_Told = thrust::raw_pointer_cast(&d_Told[0]);

			Temperature_solver<<<dimgrid, dimblock>>>(nx, ny, wu, wv, wT, dx, dy, dt, Re, Pr, ptr_u, ptr_v, ptr_Told, ptr_T);

			thrust::copy(d_Told.begin(), d_Told.end(), Told.begin());
			thrust::copy(d_T.begin(), d_T.end(), T.begin());

			int step4_end = clock();
			stepTimingAccumulator["Step 4 - Solving for temperature"] += step4_end - step4_start;
			//................................................................................................

			//...............................................................................................
			// Step i3 - Initializing boundary conditions for temperature 
			// boundary conditions for Temperature
			int stepi3_start = clock();

			for (int i = 0; i < nx + 1; i++)
			{
				for (int j = 0; j < ny + 1; j++)
				{
					if (j == 0)
					{
						T[i * wT + j] = T[i * wT + j + 1]; // bottom wall - Insulated - no flux - Final
					}
					else if (j == ny)
					{
						T[i * wT + j] = 2.0*(T_0) / T_amb - T[i * wT + j - 1]; // upper boundary - lid with ambient temperature (as air) - Final
					}
					else if (i == 0)
					{
						if (j*dy < 2.0)
						{
							T[i * wT + j] = 2.0*T_L / T_amb - T[(i + 1) * wT + j]; // left wall at T_L - Constant Temperature - Final
						}
						else
						{
							T[i * wT + j] = 2.0*T_0 / T_amb - T[(i + 1) * wT + j]; // left inlet at T_0 (initial temperature) - Final
						}
					}
					else if (i == nx)
					{
						if (j*dy < 2.0)
						{
							T[i * wT + j] = 2.0*T_R / T_amb - T[(i - 1) * wT + j]; // right wall at T_R - Final
						}
					}
				} // end for j
			} // end for i
			int stepi3_end = clock();
			stepTimingAccumulator["Step i3 - Initializing boundary conditions for temperature"] += stepi3_end - stepi3_start;
			//...............................................................................................

			//...............................................................................................
			// Step 5 - Checking if solution reached steady state
			// Checking the steady state condition
			int step5_start = clock();

			float TV, diffv; TV = 0;
			for (int i = 1; i < nx - 1; i++)
			{
				for (int j = 1; j < ny - 2; j++)
				{
					diffv = v[i * wv + j] - vold[i * wv + j];
					TV = TV + pow(pow(diffv, 2), 0.5);
				} // end for i
			} // end for j

			TV = TV / ((nx - 1)*(ny - 2));

			if (TV < st && error < eps)
			{
				cout << "Steady state time = " << t << " (s) " << endl;
				break;
			}
			counter = counter + 1;
			if (fmod(counter, 10) == 0 || counter == 1)
			{
				//cout << "" << endl;
				//cout << "Column" << setw(30) << "time(s)" << setw(30) << "Iterations on Pressure" << setw(30) << "Pressure Residual" << setw(30) << "Total Variance" << endl;
			} // end if
			int step5_end = clock();
			stepTimingAccumulator["Step 5 - Check for steady state"] += step5_end - step5_start;
			//...............................................................................................


			//cout << column << setw(30) << t << setw(30) << iter << setw(30) << error << setw(30) << TV << endl;
			g << column << setw(30) << t << setw(30) << iter << setw(30) << error << setw(30) << TV << endl;
			t = t + dt;
			column = column + 1;

		} // end while time

		//........................................................................................................

		// Step 6
		// Co-locate the staggered grid points 
		int step6_start = clock();
		for (int i = 0; i < nx; i++)
		{
			for (int j = 0; j < ny; j++)
			{
				vc[i * wc + j] = 1.0 / 2.0*(v[(i + 1) * wv + j] + v[i * wv + j]);
				pc[i * wc + j] = 1.0 / 4.0*(p[i * wp + j] + p[(i + 1) * wp + j] + p[i * wp + j + 1] + p[(i + 1) * wp + j + 1]);
				uc[i * wc + j] = 1.0 / 2.0*(u[i*wu + j] + u[i * wu + j + 1]);
				om[i * wc + j] = 1.0 / dx*(v[(i + 1) * wv + j] - v[i * wv + j]) - 1.0 / dy*(u[i * wu + j + 1] - u[i * wu + j]);
				Tc[i * wc + j] = 1.0 / 4.0*(T[i * wT + j] + T[(i + 1) * wT + j] + T[i * wT + j + 1] + T[(i + 1) * wT + j + 1]);
			} // end for j
		} // end for i
		//........................................................................................................
		int step6_end = clock();
		stepTimingAccumulator["Step 6 - Co-locate staggered grid points"] += step6_end - step6_start;

		// Steady state results

		for (int j = 0; j < ny; j++)
		{
			for (int i = 0; i < nx; i++)
			{
				f << setw(15) << t - dt << setw(15) << i*dx << setw(15) << j*dy << setw(15) << uc[i * wc + j] << setw(15) << vc[i * wc + j] << setw(15) << pc[i * wc + j] << setw(15) << Tc[i * ny + j] * T_amb - 273.15 << setw(15) << om[i * wc + j] << endl;
			} // end for i
		} // end for j
		//.........................................................................................................

		float end_clock = clock();
		cout << "CPU time = " << (end_clock - start_clock) / CLOCKS_PER_SEC << " (s)" << endl;
		//cout << "Re = " << Re << endl;
		//cout << "Fr = " << Fr << endl;

		for (auto it = stepTimingAccumulator.begin(); it != stepTimingAccumulator.end(); it++)
		{
			float seconds = (float)it->second / CLOCKS_PER_SEC;
			std::cout << it->first << "\t" << seconds << endl;
		}
	}
	catch(thrust::system_error e)
	{
		std::cerr <<  e.what() << std::endl;
	}

	return 0;
} // end main
