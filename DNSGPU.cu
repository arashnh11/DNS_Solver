#include "hip/hip_runtime.h"
#include "DNSGPU.cuh"

#include <iostream>
#include <fstream>
#include <math.h>
#include <time.h>
#include <vector>
#include <iomanip>
#include <algorithm>
#include <string>
#include <map>
#include <stdint.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/reduce.h>

#include <hip/hip_runtime_api.h>
#include <>
#include <hip/hip_runtime.h>


// Define variables in global space (bad practice but we'll go with this for now)
// Enclosed with empty namespace to avoid linking errors with the other implementation of the algorithm
namespace 
{
	using namespace std;

	DNSGPU::SIMSTATES s_nextState = DNSGPU::SIMSTATES::INIT;

	// output format
	float start_clock = clock();

	ofstream f("result_gpu.txt"); // Solution Results
	
	ofstream g("convergence_gpu.txt"); // Convergence history


	// Input parameters 
	//float Re, Pr, Fr, T_L, T_0, T_amb, ni, nj, dx, dy, t, ny, nx, eps, /*beta,*/ iter, maxiter, tf, st, counter, column, u_wind, T_R, Lx, Ly;
	
	float Lx = 4.0, Ly = 5.0; // Domain dimensions
	int ni = 2; // Number of nodes per unit length in x direction
	int nj = 2; // Number of nodes per unit length in y direction
	int nx = Lx * ni; 
	int ny = Ly * nj; // Number of Nodes in each direction
	float u_wind = 1; // Reference velocity
	float st = 0.00005; // Total variance criteria
	float eps = 0.001; // Pressure convergence criteria
	float tf = 100; // Final time step
	float Pr = 0.5*(0.709 + 0.711); // Prandtl number
	float Re = 200.0; float Fr = 0.3; // Non-dimensional numbers for inflow conditions
	float dx = Lx / (nx - 1); 
	float dy = Ly / (ny - 1); // dx and dy
											//beta = 1; // Successive over relaxation factor (SOR)
	float t = 0; // Initial time step
	float T_L = 100.0; // Left wall temperature (C)
	float T_R = 50.0; // Right wall temperature (C)
	float T_amb = 25.0; // Ambient air temperature (C)
	float T_0 = 50.0; // Initial air temperature
	
	int maxiter = 1000; // Maximum iteration at each time step
	int counter = 0; // initial row for output monitoring
	int column = 1; // Column number for output display
				// Records number of clicks a step takes
	std::map<string, uint32_t> stepTimingAccumulator;

	// Output vector (goes out to visualize data)
	std::vector<float> vizOutput;

	// Host Vectors

	thrust::host_vector<float> u(nx * (ny + 1));
	thrust::host_vector<float> us(nx*(ny + 1));
	thrust::host_vector<float> uold(nx * (ny + 1));
	int wu = ny + 1;

	thrust::host_vector<float> v((nx + 1) * ny);
	thrust::host_vector<float> vs((nx + 1) * ny);
	thrust::host_vector<float> vold((nx + 1) * ny);
	int wv = ny;

	thrust::host_vector<float> p((nx + 1) * (ny + 1));
	int wp = ny + 1;



	thrust::host_vector<float> T;
	int wT = ny + 1;

	// These are initialized in the Init() function
	thrust::host_vector<float> Told;
	thrust::host_vector<float> om;
	thrust::host_vector<float> vc;
	thrust::host_vector<float> uc;
	thrust::host_vector<float> pc;
	thrust::host_vector<float> Tc;
	int wc = ny;

	thrust::device_vector<float> us_d(nx*(ny + 1));
	thrust::device_vector<float> vs_d((nx + 1) * ny);
	thrust::device_vector<float> p_d((nx + 1) * (ny + 1));
	thrust::device_vector<float> p_old((nx + 1) * (ny + 1));
	thrust::device_vector<float> p_ref((nx + 1) * (ny + 1));
	thrust::device_vector<float> abs_d((nx + 1) * (ny + 1));

#define BLOCK_SIZE 32 // Number of threads in x and y direction - Maximum Number of threads per block = 32 * 32 = 1024

	// Time step size stability criterion

	float mt1 = 0.25*pow(dx, 2.0) / (1.0 / Re); float Rer = 1.0 / Re; float mt2 = 0.25*pow(dy, 2.0) / (1.0 / Re);
	float dt;

}


void DNSGPU::RunSimulation()
{
	switch (s_nextState)
	{
	case INIT: Init(); break;
	case STEP: Step(); break;
	case FINISH: Finish(); break;
	case END: break;
	}


}

// Call this to initialize shit
void DNSGPU::Init()
{
	f.setf(ios::fixed | ios::showpoint);
	f << setprecision(5);

	g.setf(ios::fixed | ios::showpoint);
	g << setprecision(5);
	cout.setf(ios::fixed | ios::showpoint);
	cout << setprecision(5);

	T_L = T_L + 273.15; T_0 = T_0 + 273.15; T_amb = T_amb + 273.15; T_R = T_R + 273.15;// Unit conversion to (K)

	T.resize((nx + 1) * (ny + 1), T_0 / T_amb);
	Told.resize((nx + 1) * (ny + 1), 0);
	om.resize(nx * ny, 0);
	vc.resize(nx * ny, 0);
	uc.resize(nx * ny, 0);
	pc.resize(nx * ny, 0);
	Tc.resize(nx*ny, 0);

	if (mt1 > Rer)
	{
		dt = Rer;
	}
	else
	{
		dt = mt1;
	}

	if (dt > mt2)
	{
		dt = mt2;
	}
	cout << "dt = " << dt << endl;

	s_nextState = STEP;
}

__global__ void Temperature_solver(int nx, int ny, int wu, int wv, int wT, float dx, float dy, float dt, float Re, float Pr, float *u, float *v, float *Told, float *T)
{

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if (i > 0 && i < nx && j > 0 && j < ny) {

		Told[i * wT + j] = T[i * wT + j];
		T[i * wT + j] = T[i * wT + j] + dt*(-0.5*(u[i * wu + j] + u[(i - 1) * wu + j])*(1.0 / (2.0*dx)*(T[(i + 1) * wT + j] - T[(i - 1) * wT + j])) - 0.5*(v[i * wv + j] + v[i * wv + j - 1])*(1.0 / (2.0*dy)*(T[i * wT + j + 1] - T[i * wT + j - 1])) + 1 / (Re*Pr)*(1 / pow(dx, 2.0f)*(T[(i + 1) * wT + j] - 2.0*T[i * wT + j] + T[(i - 1) * wT + j]) + 1 / pow(dy, 2.0f)*(T[i * wT + j + 1] - 2 * T[i * wT + j] + T[i * wT + j - 1])));
	}
	__syncthreads();
}



__global__ void PressureSolve(float * p_d, const float * p_old, float * abs_d, const float * us_d, const float * vs_d, int p_xlength, int p_ylength, int wp, int wu, int wv, float dx, float dy, float dt)
{

	int i = threadIdx.x + blockDim.x*blockIdx.x;
	int j = threadIdx.y + blockDim.y*blockIdx.y;


	if (i > 0 && i < p_xlength && j > 0 && j < p_ylength)
	{
		//        __syncthreads();

		p_d[i * wp + j] = pow(dx, 2.0f)*pow(dy, 2.0f) / (-2.0*(pow(dx, 2.0f) + pow(dy, 2.0f)))*(-1.0 / pow(dx, 2.0f)*(p_old[(i + 1) * wp + j] + p_old[(i - 1) * wp + j] + p_old[i * wp + j + 1] + p_old[i * wp + j - 1]) + 1.0 / dt*(1.0 / dx*(us_d[i * wu + j] - us_d[(i - 1) * wu + j]) + 1.0 / dy*(vs_d[i * wv + j] - vs_d[i * wv + j - 1])));
		__syncthreads();

		abs_d[i * wp + j] = p_d[i * wp + j] - p_old[i * wp + j];

		__syncthreads();

		abs_d[i * wp + j] = abs_d[i * wp + j] * abs_d[i * wp + j];
		//__syncthreads();
	} // end if


} // end global


__global__ void PressureBC(float * p_d, float * p_ref, int nx, int ny, float dy, int wp)
{

	int i = threadIdx.x + blockDim.x*blockIdx.x;
	int j = threadIdx.y + blockDim.y*blockIdx.y;

	if (i >= 0 && i < nx + 1 && j == 0) {
		p_d[i * wp + j] = p_ref[i * wp + j + 1]; // bottom wall - Final
	}
	__syncthreads();
	if (i >= 0 && i < nx + 1 && j == ny) {
		p_d[i * wp + j] = p_ref[i * wp + j - 1]; // Upper - no flux
	}
	__syncthreads();
	if (j >= 0 && j < ny + 1 && i == 0) {
		p_d[i * wp + j] = p_ref[(i + 1) * wp + j]; // left wall - not the inlet - Final
	}
	__syncthreads();
	if (j >= 0 && j < ny + 1 && i == nx && j*dy < 2.0) {
		p_d[i * wp + j] = p_ref[(i - 1) * wp + j]; // right wall - not the outlet - Final

												   // printf("POSITIVE ");
	}
	__syncthreads();
	if (j >= 0 && j < ny + 1 && i == nx && j*dy >= 2.0) {
		p_d[i * wp + j] = -p_ref[(i - 1) * wp + j]; // pressure outlet - static pressure is zero - Final
													// printf("NEGATIVE ");    

	}
	//__syncthreads();

}

void DNSGPU::Step()
{
	hipDeviceSynchronize();
	// dt is defined statically 
	if(t<tf)
	{
		int stepi1_start = clock();
		//........................................................................................
		// Step i1 - it can be parallelized 
		// boundary conditions for u velocity

		for (int i = 0; i < nx; i++)
		{
			for (int j = 0; j < ny + 1; j++)
			{
				if (i == 0 && j > 0 && j < ny)
				{
					if (j*dy < 2.0)
					{
						u[i * wu + j] = 0; // left wall - Final
					}
					else
					{
						u[i * wu + j] = u_wind; // left inlet - Final
					}
				}
				else if (i == nx - 1 && j>0 && j < ny)
				{
					if (j*dy < 2.0)
					{
						u[i * wu + j] = 0; // Right wall has 0 horizontal velocity - Final
					}
					else
					{
						u[i * wu + j] = u[(i - 1) * wu + j]; // right outlet - no velocity change
					}
				}
				else if (j == 0)
				{
					u[i * wu + j] = -u[i * wu + j + 1]; // bottom ghost - Final
				}
				else if (j == ny)
				{
					u[i * wu + j] = u[i * wu + j - 1]; // upper ghost - Final
				}
			} // end for j
		} // end for i
		int stepi1_end = clock();
		stepTimingAccumulator["Step i1 - Set Horizontal Velocity Boundary Conditions"] += stepi1_end - stepi1_start;
		//...............................................................................................


		//.........................................................................................
		// Step i2 - it can be parallelized
		// boundary conditions for v velocity
		int stepi2_start = clock();

		for (int i = 0; i < nx + 1; i++)
		{
			for (int j = 0; j < ny; j++)
			{
				if (j == 0 && i > 0 && i < nx)
				{
					v[i * wv + j] = 0; // bottom wall - Final
				}
				else if (j == ny - 1 && i > 0 && i < nx)
				{
					v[i * wv + j] = v[i * wv + j - 1]; // upper wall - Final
				}
				else if (i == 0)
				{
					v[i * wv + j] = -v[(i + 1) * wv + j]; // left ghost (Left Wall and inlet has 0 vertical velocity) - Final
				}
				else if (i == nx)
				{
					if (j*dy < 2.0)
					{
						v[i * wv + j] = -v[(i - 1) * wv + j]; // right ghost (Right wall has 0 vertical velocity) - Final
					}
					else
					{
						v[i * wv + j] = v[(i - 1) * wv + j]; // right outlet - no velocity gradient
					}
				}
			} // end for j
		} // end for I
		int stepi2_end = clock();
		stepTimingAccumulator["Step i2 - Set Vertical Velocity Boundary Conditions"] += stepi2_end - stepi2_start;
		//...............................................................................................

		//...............................................................................................
		int step1_start = clock();
		//.........................................................................................
		// Step 1 - it can be parallelized - Solve for intermediate velocity values

		// u - us - vh - a 

		for (int i = 1; i < nx - 1; i++)
		{
			for (int j = 1; j < ny; j++)
			{
				float vh = 1.0 / 4.0*(v[i * wv + j] + v[(i + 1) * wv + j] + v[i * wv + j - 1] + v[(i + 1) * wv + j - 1]); // v hat
				float a = u[i * wu + j] * 1.0 / (2.0*dx)*(u[(i + 1) * wu + j] - u[(i - 1) * wu + j]) + vh*1.0 / (2.0*dy)*(u[i * wu + j + 1] - u[i * wu + j - 1]); // a
				us[i * wu + j] = dt / Re*(1.0 / pow(dx, 2.0)*(u[(i + 1) * wu + j] - 2.0*u[i * wu + j] + u[(i - 1) * wu + j]) + 1.0 / pow(dy, 2.0)*(u[i * wu + j + 1] - 2.0*u[i * wu + j] + u[i * wu + j - 1])) - a*dt + u[i * wu + j]; // u star
			} // end for j
		} // end for i

		  //..........................................................................................
		  // Step 1 - it can be parallelized
		  // v - vs - uh - b
		for (int i = 1; i < nx; i++)
		{
			for (int j = 1; j < ny - 1; j++)
			{
				float uh = 1.0 / 4.0*(u[i * wu + j] + u[i * wu + j + 1] + u[(i - 1) * wu + j] + u[(i - 1) * wu + j + 1]);
				float b = uh*1.0 / (2.0*dx)*(v[(i + 1) * wv + j] - v[(i - 1) * wv + j]) + v[i * wv + j] * 1.0 / (2.0*dy)*(v[i * wv + j + 1] - v[i * wv + j - 1]); // b
				vs[i * wv + j] = dt / Re*(1.0 / pow(dx, 2.0)*(v[(i + 1) * wv + j] - 2.0*v[i * wv + j] + v[(i - 1) * wv + j]) + 1.0 / pow(dy, 2.0)*(v[i * wv + j + 1] - 2.0*v[i * wv + j] + v[i * wv + j - 1])) + dt / pow(Fr, 2.0)*(0.5*(T[i * wT + j] + T[i * wT + j - 1]) - 1) / (0.5*(T[i * wT + j] + T[i * wT + j - 1])) - b*dt + v[i * wv + j]; // v 
			} // end for j
		} // end for i

		  //...........................................................................................
		  // vs and us on Boundary conditions

		for (int i = 0; i < nx; i++)
		{
			us[i * wu + 0] = -us[i * wu + 1]; // bottom ghost - Final
		} // end for j

		  //...........................................................................................
		for (int j = 0; j < ny + 1; j++)
		{
			if (j*dy < 2.0)
			{
				us[0 * wu + j] = 0; // left wall - FInal
				us[(nx - 1) * wu + j] = 0; // right wall - Final
			}
			else
			{
				us[0 * wu + j] = u_wind; // left inlet - Final
			}
		}
		//...........................................................................................

		for (int j = 0; j < ny; j++)
		{
			vs[0 * wv + j] = -vs[1 * wv + j]; // left ghost (Both wall and inlet have 0 vs) - Final
			if (j*dy < 2.0)
			{
				vs[nx * wv + j] = -vs[(nx - 1) * wv + j]; // right ghost (Only the right wall - Final
			}
			else
			{
				vs[nx * wv + j] = vs[(nx - 1) * wv + j]; // right outlet - no flux
			}
		}
		//............................................................................................

		for (int i = 0; i < nx + 1; i++)
		{
			vs[i * wv + 0] = 0; // Bottom wall - Final
		} // end for i
		  //............................................................................................

		int step1_end = clock();
		stepTimingAccumulator["Step 1 - Solve for intermediate velocities"] += step1_end - step1_start;

		//...............................................................................................
		// Step 2 - It can be parallelized 
		// This is the most expensive part of the code
		// Poisson equation for pressure
		int step2_start = clock();
		// Cuda set up
		int p_xlength = nx;
		int p_ylength = ny;

		float *ptr_us = thrust::raw_pointer_cast(&us_d[0]);
		float *ptr_vs = thrust::raw_pointer_cast(&vs_d[0]);
		float *ptr_p = thrust::raw_pointer_cast(&p_d[0]);
		float *ptr_p_old = thrust::raw_pointer_cast(&p_old[0]);
		float *ptr_abs = thrust::raw_pointer_cast(&abs_d[0]);
		float *ptr_p_ref = thrust::raw_pointer_cast(&p_ref[0]);

		float error = 1.0; int iter = 0;
		// float diffp = 0;
		us_d = us;
		vs_d = vs;

		// Begin Jacobi loop
		while (error > eps && iter < maxiter) {
			p_old = p_d;

			// Jacobi pressure solver
			PressureSolve <<< dim3((ny + 1) / BLOCK_SIZE + 1, (nx + 1) / BLOCK_SIZE + 1, 1), dim3(BLOCK_SIZE, BLOCK_SIZE, 1) >> >(ptr_p, ptr_p_old, ptr_abs, ptr_us, ptr_vs, p_xlength, p_ylength, wp, wu, wv, dx, dy, dt);
			hipDeviceSynchronize();

			p_ref = p_d;

			error = thrust::reduce(abs_d.begin(), abs_d.end());

			PressureBC <<< dim3((ny + 1) / BLOCK_SIZE + 1, (nx + 1) / BLOCK_SIZE + 1, 1), dim3(BLOCK_SIZE, BLOCK_SIZE, 1) >> >(ptr_p, ptr_p_ref, nx, ny, dy, wp);

			hipDeviceSynchronize();

			error = pow(error, 0.5);
			iter = iter + 1;

		} // end while eps

		p = p_d;

		int step2_end = clock();
		stepTimingAccumulator["Step 2 - Solve for pressure until tolerance or max iterations"] += step2_end - step2_start;
		//...............................................................................................

		//.................................................................................................
		// Step 3 - It can be parallelized 
		// velocity update - projection method
		int step3_start = clock();

		// u

		for (int i = 1; i < nx - 1; i++)
		{
			for (int j = 1; j < ny; j++)
			{
				uold[i * wu + j] = u[i * wu + j];
				u[i * wu + j] = us[i * wu + j] - dt / dx*(p[(i + 1) * wp + j] - p[i * wp + j]);
			} // end for j
		} // end for i
		  //................................................

		  // v

		for (int i = 1; i < nx; i++)
		{
			for (int j = 1; j < ny - 1; j++)
			{
				vold[i * wv + j] = v[i * wv + j];
				v[i * wv + j] = vs[i * wv + j] - dt / dy*(p[i * wp + j + 1] - p[i * wp + j]);
			} // end for j
		} // end for i
		int step3_end = clock();
		stepTimingAccumulator["Step 3 - Velocity Update"] += step3_end - step3_start;
		//...............................................................................................

		//...............................................................................................
		// Step 4 - It can be parallelized
		// Solving for temperature
		int step4_start = clock();

		thrust::device_vector<float> d_T = T;
		thrust::device_vector<float> d_Told = Told;
		thrust::device_vector<float> d_u = u;
		thrust::device_vector<float> d_v = v;

		int gridsize_x = nx / BLOCK_SIZE + 1;
		int gridsize_y = ny / BLOCK_SIZE + 1;

		dim3 dimgrid(gridsize_x, gridsize_y, 1); // The grid has #gridsize blocks in x and 1 block in y and 1 block in z direction
		dim3 dimblock(BLOCK_SIZE, BLOCK_SIZE, 1);

		float *ptr_u = thrust::raw_pointer_cast(&d_u[0]);
		float *ptr_v = thrust::raw_pointer_cast(&d_v[0]);
		float *ptr_T = thrust::raw_pointer_cast(&d_T[0]);
		float *ptr_Told = thrust::raw_pointer_cast(&d_Told[0]);

		Temperature_solver << <dimgrid, dimblock >> >(nx, ny, wu, wv, wT, dx, dy, dt, Re, Pr, ptr_u, ptr_v, ptr_Told, ptr_T);

		thrust::copy(d_Told.begin(), d_Told.end(), Told.begin());
		thrust::copy(d_T.begin(), d_T.end(), T.begin());

		int step4_end = clock();
		stepTimingAccumulator["Step 4 - Solving for temperature"] += step4_end - step4_start;
		//................................................................................................

		//...............................................................................................
		// Step i3 - Initializing boundary conditions for temperature 
		// boundary conditions for Temperature
		int stepi3_start = clock();

		for (int i = 0; i < nx + 1; i++)
		{
			for (int j = 0; j < ny + 1; j++)
			{
				if (j == 0)
				{
					T[i * wT + j] = T[i * wT + j + 1]; // bottom wall - Insulated - no flux - Final
				}
				else if (j == ny)
				{
					T[i * wT + j] = 2.0*(T_0) / T_amb - T[i * wT + j - 1]; // upper boundary - lid with ambient temperature (as air) - Final
				}
				else if (i == 0)
				{
					if (j*dy < 2.0)
					{
						T[i * wT + j] = 2.0*T_L / T_amb - T[(i + 1) * wT + j]; // left wall at T_L - Constant Temperature - Final
					}
					else
					{
						T[i * wT + j] = 2.0*T_0 / T_amb - T[(i + 1) * wT + j]; // left inlet at T_0 (initial temperature) - Final
					}
				}
				else if (i == nx)
				{
					if (j*dy < 2.0)
					{
						T[i * wT + j] = 2.0*T_R / T_amb - T[(i - 1) * wT + j]; // right wall at T_R - Final
					}
				}
			} // end for j
		} // end for i
		int stepi3_end = clock();
		stepTimingAccumulator["Step i3 - Initializing boundary conditions for temperature"] += stepi3_end - stepi3_start;
		//...............................................................................................

		//...............................................................................................
		// Step 5 - Checking if solution reached steady state
		// Checking the steady state condition
		int step5_start = clock();

		float TV, diffv; TV = 0;
		for (int i = 1; i < nx - 1; i++)
		{
			for (int j = 1; j < ny - 2; j++)
			{
				diffv = v[i * wv + j] - vold[i * wv + j];
				TV = TV + pow(pow(diffv, 2), 0.5);
			} // end for i
		} // end for j

		TV = TV / ((nx - 1)*(ny - 2));

		if (TV < st && error < eps)
		{
			cout << "Steady state time = " << t << " (s) " << endl;
			s_nextState = FINISH;
			return;
		}
		counter = counter + 1;
		if (fmod(counter, 10) == 0 || counter == 1)
		{
			//cout << "" << endl;
			//cout << "Column" << setw(30) << "time(s)" << setw(30) << "Iterations on Pressure" << setw(30) << "Pressure Residual" << setw(30) << "Total Variance" << endl;
		} // end if
		int step5_end = clock();
		stepTimingAccumulator["Step 5 - Check for steady state"] += step5_end - step5_start;
		//...............................................................................................


		//cout << column << setw(30) << t << setw(30) << iter << setw(30) << error << setw(30) << TV << endl;
		g << column << setw(30) << t << setw(30) << iter << setw(30) << error << setw(30) << TV << endl;
		t = t + dt;
		column = column + 1;
	}
	else
	{
		s_nextState = FINISH;
	}
}

void DNSGPU::Finish()
{
	// Step 6
	// Co-locate the staggered grid points 
	int step6_start = clock();
	for (int i = 0; i < nx; i++)
	{
		for (int j = 0; j < ny; j++)
		{
			vc[i * wc + j] = 1.0 / 2.0*(v[(i + 1) * wv + j] + v[i * wv + j]);
			pc[i * wc + j] = 1.0 / 4.0*(p[i * wp + j] + p[(i + 1) * wp + j] + p[i * wp + j + 1] + p[(i + 1) * wp + j + 1]);
			uc[i * wc + j] = 1.0 / 2.0*(u[i*wu + j] + u[i * wu + j + 1]);
			om[i * wc + j] = 1.0 / dx*(v[(i + 1) * wv + j] - v[i * wv + j]) - 1.0 / dy*(u[i * wu + j + 1] - u[i * wu + j]);
			Tc[i * wc + j] = 1.0 / 4.0*(T[i * wT + j] + T[(i + 1) * wT + j] + T[i * wT + j + 1] + T[(i + 1) * wT + j + 1]);
		} // end for j
	} // end for i
	  //........................................................................................................
	int step6_end = clock();
	stepTimingAccumulator["Step 6 - Co-locate staggered grid points"] += step6_end - step6_start;

	// Steady state results

	for (int j = 0; j < ny; j++)
	{
		for (int i = 0; i < nx; i++)
		{
			f << setw(15) << t - dt << setw(15) << i*dx << setw(15) << j*dy << setw(15) << uc[i * wc + j] << setw(15) << vc[i * wc + j] << setw(15) << pc[i * wc + j] << setw(15) << Tc[i * ny + j] * T_amb - 273.15 << setw(15) << om[i * wc + j] << endl;
		} // end for i
	} // end for j
	  //.........................................................................................................

	float end_clock = clock();
	cout << "CPU time = " << (end_clock - start_clock) / CLOCKS_PER_SEC << " (s)" << endl;
	//cout << "Re = " << Re << endl;
	//cout << "Fr = " << Fr << endl;

	for (auto it = stepTimingAccumulator.begin(); it != stepTimingAccumulator.end(); it++)
	{
		float seconds = (float)it->second / CLOCKS_PER_SEC;
		std::cout << it->first << "\t" << seconds << endl;
	}

	s_nextState = END;
}
DNSGPU::DNSGPU()
{
}


DNSGPU::~DNSGPU()
{
}


std::vector<float>& DNSGPU::getPressure()
{
	vizOutput.resize(p.size());
	for (int i = 0; i != p.size(); ++i)
		vizOutput[i] = p[i];

	return vizOutput;
}
float DNSGPU::getPressureWidth()
{
	return wp;
}

std::vector<float>& DNSGPU::getTemperature()
{
	vizOutput.resize(T.size());

	for (int i = 0; i != T.size(); ++i)
		vizOutput[i] = T[i];

	return vizOutput;
}
float DNSGPU::getTemperatureWidth()
{
	return wT;
}

std::vector<float>& DNSGPU::getU()
{
	vizOutput.resize(u.size());

	for (int i = 0; i != u.size(); ++i)
		vizOutput[i] = u[i];

	return vizOutput;
}
float DNSGPU::getUWidth()
{
	return wu;
}
